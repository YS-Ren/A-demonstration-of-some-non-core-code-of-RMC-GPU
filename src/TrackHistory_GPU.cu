#include "NeutronTracking_GPU.h"
#include "Criticality_GPU.h"
#include "Timers_GPU.h"
#include"EventControl_GPU.h"
#include"Tally_GPU.h"

void TrackHistory_GPUh(int threadNum) {
    int numBlocks = (threadNum + blockSize - 1) / blockSize;

    int NeuNumAlive = threadNum;
    hipMemcpyToSymbol(HIP_SYMBOL(NeuNumAlive_GPU), &NeuNumAlive, sizeof(int));
    int zero = 0;
    int h_NeuCrossNum = threadNum;
    int h_NeuColliNum = 0;
    int iter = 0;
    RayTracking_Init_GPUg << <numBlocks, blockSize >> > (threadNum);  hipDeviceSynchronize();
    for (;;) {
        /// - ��ʼ��������������ײ������
        hipMemcpyToSymbol(HIP_SYMBOL(ParticlesEventState_GPU), &zero, sizeof(int), offsetof(CDParticlesEventState_GPU, NeuCrossNum));
        hipMemcpyToSymbol(HIP_SYMBOL(ParticlesEventState_GPU), &zero, sizeof(int), offsetof(CDParticlesEventState_GPU, NeuColliNum));

        /// - ������
        if (h_NeuCrossNum != 0) {
            numBlocks = (h_NeuCrossNum + blockSize - 1) / blockSize;
            RayTracking_GPUg << <numBlocks, blockSize >> > (h_NeuCrossNum); hipDeviceSynchronize();
        }
        /// - ������ײ\n
        if (h_NeuColliNum != 0) {
            numBlocks = (h_NeuColliNum + blockSize - 1) / blockSize;
            TreatColli_GPUg << <numBlocks, blockSize >> > (h_NeuColliNum);
        }
        hipDeviceSynchronize();
        numBlocks = (threadNum + blockSize - 1) / blockSize;
        CountAliveNeuNum_GPUg << <numBlocks, blockSize >> > (threadNum);
        InterchangePSEventPtr_GPUg << <1, 1 >> > (); hipDeviceSynchronize();
        hipMemcpyFromSymbol(&NeuNumAlive, HIP_SYMBOL(NeuNumAlive_GPU), sizeof(int));

        if (NeuNumAlive <= NumOfSP) {
            break;
        }

        /// - ��ȡ������������ײ������
        hipMemcpyFromSymbol(&h_NeuCrossNum, HIP_SYMBOL(ParticlesEventState_GPU), sizeof(int), offsetof(CDParticlesEventState_GPU, NeuCrossNum));
        hipMemcpyFromSymbol(&h_NeuColliNum, HIP_SYMBOL(ParticlesEventState_GPU), sizeof(int), offsetof(CDParticlesEventState_GPU, NeuColliNum));

        iter++;
    }
    InterchangePSEventPtr_Cross_GPUg << <1, 1 >> > (); hipDeviceSynchronize();
    hipMemcpyFromSymbol(&h_NeuColliNum, HIP_SYMBOL(ParticlesEventState_GPU), sizeof(int), offsetof(CDParticlesEventState_GPU, NeuColliNum));
    if (h_NeuColliNum != 0) {
        numBlocks = (h_NeuColliNum + blockSize - 1) / blockSize;
        TreatColli_GPUg << <numBlocks, blockSize >> > (h_NeuColliNum);
        hipDeviceSynchronize();
    }
    InterchangePSEventPtr_Cross_GPUg << <1, 1 >> > (); hipDeviceSynchronize();
    hipMemcpyFromSymbol(&NeuNumAlive, HIP_SYMBOL(NeuNumAlive_GPU), sizeof(int));
    hipMemcpyFromSymbol(&h_NeuCrossNum, HIP_SYMBOL(ParticlesEventState_GPU), sizeof(int), offsetof(CDParticlesEventState_GPU, NeuCrossNum));
    hipMemcpyToSymbol(HIP_SYMBOL(ParticlesEventState_GPU), &zero, sizeof(int), offsetof(CDParticlesEventState_GPU, NeuCrossNum));
    hipMemcpyToSymbol(HIP_SYMBOL(ParticlesEventState_GPU), &zero, sizeof(int), offsetof(CDParticlesEventState_GPU, NeuColliNum));
    if (h_NeuCrossNum != 0) {
        numBlocks = (h_NeuCrossNum + blockSize - 1) / blockSize;
        TrackHistoryAfterEvent_GPUg << <numBlocks, blockSize >> > (h_NeuCrossNum);
        hipDeviceSynchronize();
    }
}
