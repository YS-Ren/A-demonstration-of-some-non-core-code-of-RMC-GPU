#include "hip/hip_runtime.h"
#include "GPUErrorCtrl.h"

#include <iostream>
#include <fstream>

#include "GPUPrint.h"

//**************************************************
//	class def
//**************************************************
// class CDGPULog
CDGPULog::CDGPULog(size_t capacity) {
    initialize(capacity);
}

// ��ʼ����־����ջ�������
void CDGPULog::initialize(size_t capacity) {
    _capacity = capacity;
    _used = 0;
    _isFull = false;
    hipMalloc(reinterpret_cast<void**>(&_log), capacity);
}

// �豸�˼�¼������Ϣ
__device__ size_t CDGPULog::record(const char* error_str) {
    // �����ַ������ȣ���������ֹ����
    size_t len = 0;
    while (error_str[len] != '\0') len++;

    // ԭ���Ե�������ʹ���ֽ���
    size_t start_idx = atomicAdd(&_used, len + 1);  // +1 ���ڻ��з�

    // ����Ƿ����㹻�ռ�
    if (start_idx + len + 1 >= _capacity) {
        _isFull = true;
        return 0;  // �ռ䲻�㣬����¼
    }

    // �����ַ�������־������
    for (size_t i = 0; i < len; i++) {
        _log[start_idx + i] = error_str[i];
    }

    // ��ӻ��з�
    _log[start_idx + len] = '\n';
    return start_idx;
}

// ���豸�˸��ƴ������ݵ�������
void CDGPULog::copyFromDev(std::vector<char>& h_ErrorLog) {
    // ������ʱ������
    h_ErrorLog.resize(_capacity);

    // ���豸�˸�������
    hipMemcpy(h_ErrorLog.data(), _log, _capacity * sizeof(char), hipMemcpyDeviceToHost);
}

// ��ӡ������־���ļ�
//void CDGPULog::printToFile(const char* filename) {
//
//    if (_used == 0) {
//        printf("No errors logged.\n");
//        return;
//    }
//
//    // ���������˻������������豸����
//    char* host_log = new char[_used + 1];
//    hipMemcpy(host_log, _log, _used, hipMemcpyDeviceToHost);
//    host_log[_used] = '\0';
//
//    // д���ļ�
//    FILE* file = fopen(filename, "w");
//    if (file) {
//        fwrite(host_log, 1, _used, file);
//        fclose(file);
//        printf("Error log written to %s\n", filename);
//    }
//    else {
//        printf("Failed to open file %s for writing\n", filename);
//    }
//    delete[] host_log;
//}

std::string CDGPULog::getLog(size_t startPos, char* log) {
    if (log == nullptr || startPos >= strlen(log)) {
        return "";
    }

    char* startPtr = log + startPos;
    char* newlinePtr = strchr(startPtr, '\n');

    return (newlinePtr != nullptr)
        ? std::string(startPtr, newlinePtr)
        : std::string(startPtr);
}


//class CDGPUErrorCtrl
// ��ʼ������
CDGPUErrorCtrl::CDGPUErrorCtrl(size_t ErrorCapacity, size_t logCapacity) {
    initialize(ErrorCapacity, logCapacity);
}

void CDGPUErrorCtrl::initialize(size_t ErrorCapacity, size_t logCapacity) {
    _errorCapacity = ErrorCapacity;
    _errorCount = 0;
    _errorCyc = 0;
    // �����豸�ڴ�
    hipMalloc(&_errorCodes, _errorCapacity * sizeof(int));
    hipMalloc(&_errorParticleIds, _errorCapacity * sizeof(size_t));
    hipMalloc(&_errorLogStartPos, _errorCapacity * sizeof(size_t));

    // ��ʼ���豸�ڴ�
    hipMemset(_errorCodes, 0, _errorCapacity * sizeof(int));
    hipMemset(_errorParticleIds, 0, _errorCapacity * sizeof(size_t));
    hipMemset(_errorLogStartPos, 0, _errorCapacity * sizeof(size_t));

    _errorLog.initialize(logCapacity);
}

// ���豸�˸��ƴ������ݵ�������
void CDGPUErrorCtrl::copyFromDev(std::vector<int>& h_ErrorCode, std::vector<size_t>& h_ErrorThread, std::vector<size_t>& h_ErrorLogStartPos) {
    // ���Ʋ�������������
    _errorCount = (_errorCount > _errorCapacity) ? _errorCapacity : _errorCount;

    if (_errorCount > 0) {
        // ������ʱ������
        h_ErrorCode.resize(_errorCount);
        h_ErrorThread.resize(_errorCount);
        h_ErrorLogStartPos.resize(_errorCount);

        // ���豸�˸�������
        hipMemcpy(h_ErrorCode.data(), _errorCodes, _errorCount * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_ErrorThread.data(), _errorParticleIds, _errorCount * sizeof(size_t), hipMemcpyDeviceToHost);
        hipMemcpy(h_ErrorLogStartPos.data(), _errorLogStartPos, _errorCount * sizeof(size_t), hipMemcpyDeviceToHost);
    }
}

// �����豸�˴����¼
void CDGPUErrorCtrl::reset() {
    _errorCount = 0;
    _errorCyc = 0;
    hipMemset(_errorCodes, 0, _errorCapacity * sizeof(int));
    hipMemset(_errorParticleIds, 0, _errorCapacity * sizeof(size_t));
    hipMemset(_errorLogStartPos, 0, _errorCapacity * sizeof(size_t));
    _errorLog.reset();
}

// ��ӡ������Ϣ
void CDGPUErrorCtrl::print(const std::string& filename) {
    if (!isErrorOccur()) {
        std::stringstream ss;
        ss << "No errors recorded.";
        OLogger.info(ss.str(), "CDErrorCtrl");
        return;
    }

    // �����˴����¼����(���ڴ��豸�˸�������)
    std::vector<int> h_ErrorCode;
    std::vector<size_t> h_ErrorThread;
    std::vector<size_t> h_ErrorLogStartPos;
    copyFromDev(h_ErrorCode, h_ErrorThread, h_ErrorLogStartPos);

    std::vector<char> h_ErrorLog;
    _errorLog.copyFromDev(h_ErrorLog);

    // write to file
    std::ofstream outFile(filename);
    if (!outFile.is_open()) {
        std::stringstream ss;
        ss << "Failed to open error log file: " << filename;
        OLogger.error(ss.str());
        return;
    }

    for (size_t i = 0; i < h_ErrorCode.size(); ++i) {
        outFile << "Error["<<i<<"] [ParticleId] " << h_ErrorThread[i] << " \t[Type] ";
        // ʹ��ӳ�����Ҷ�Ӧ���ַ���
        eGPUErrorType error = static_cast<eGPUErrorType>(h_ErrorCode[i]);
        auto it = errorToString.find(error);
        outFile << it->second.c_str();
        std::string log = _errorLog.getLog(h_ErrorLogStartPos[i], h_ErrorLog.data());
        outFile << " [Position] " << log.c_str() << std::endl;
    }

    outFile.close();
    {
        std::stringstream ss;
        ss << "Error log written to: " << filename;
        OLogger.info(ss.str(), "CDErrorCtrl");
    }
    
    // print summary
    {
        std::stringstream ss;
        ss << "Error Summary (" << h_ErrorCode.size() << " errors):";
        OLogger.header(ss.str(), "CDErrorCtrl");
    }
    for (size_t i = 0; i < std::min(h_ErrorCode.size(), static_cast<size_t>(5)); ++i) {
        std::stringstream ss;
        ss << "Error[" << i << "], [ParticleId] " << h_ErrorThread[i] << " [Type] ";
        // ʹ��ӳ�����Ҷ�Ӧ���ַ���
        eGPUErrorType error = static_cast<eGPUErrorType>(h_ErrorCode[i]);
        auto it = errorToString.find(error);
        ss << it->second.c_str();
        OLogger.main(ss.str());
    }

    if (h_ErrorCode.size() > 5) {
        std::stringstream ss;
        ss << "  ... and " << (h_ErrorCode.size() - 5) << " more errors.";
        OLogger.main(ss.str());
    }
}

// �豸�˴����¼����
__device__ void CDGPUErrorCtrl::record(eGPUErrorType errorCode, size_t particleId, const char* log) {

    size_t startPos = _errorLog.record(log);
    
    // ʹ��ԭ�Ӳ�����ȡ�����¼λ��
    size_t index = atomicAdd(&_errorCount, 1);
    _errorLogStartPos[index] = startPos;

    if ((index < _errorCapacity) && (!_errorLog.isFull())) {
        // ��¼���������߳�ID
        _errorCodes[index] = static_cast<int>(errorCode);
        _errorParticleIds[index] = particleId;
    }
}

__device__ void strMergeDevice(char* output, int outputSize,
    const char* cstr, const char* filename, int line) {
    // �ֶ�ʵ�ָ�ʽ��������ʹ�ñ�׼��
    int pos = 0;

    // �����ļ���
    const char* f = filename;
    while (*f && pos < outputSize - 1) {
        output[pos++] = *f++;
    }

    // ���ð��
    if (pos < outputSize - 1) output[pos++] = ':';

    // ����кţ��ֶ�ת���������ַ�����
    int num = line;
    char numStr[16];
    int numLen = 0;

    // �����к�Ϊ0�����
    if (num == 0) {
        numStr[numLen++] = '0';
    }
    else {
        // ת���������ַ���
        int temp = num;
        while (temp > 0) {
            numLen++;
            temp /= 10;
        }
        temp = num;
        for (int i = numLen - 1; i >= 0; i--) {
            numStr[i] = '0' + (temp % 10);
            temp /= 10;
        }
    }

    // ����к��ַ���
    for (int i = 0; i < numLen && pos < outputSize - 1; i++) {
        output[pos++] = numStr[i];
    }

    // ��ӵ�Ϳո�
    if (pos < outputSize - 1) output[pos++] = ' ';
    if (pos < outputSize - 1) output[pos++] = '[';
    if (pos < outputSize - 1) output[pos++] = 'D';
    if (pos < outputSize - 1) output[pos++] = 'e';
    if (pos < outputSize - 1) output[pos++] = 't';
    if (pos < outputSize - 1) output[pos++] = 'a';
    if (pos < outputSize - 1) output[pos++] = 'i';
    if (pos < outputSize - 1) output[pos++] = 'l';
    if (pos < outputSize - 1) output[pos++] = ']';
    if (pos < outputSize - 1) output[pos++] = ' ';

    // ���ԭʼ�ַ���
    const char* s = cstr;
    while (*s && pos < outputSize - 1) {
        output[pos++] = *s++;
    }

    // ����ַ���������
    output[pos] = '\0';
}

//**************************************************
//	extern var
//**************************************************

__device__ CDGPUErrorCtrl OGPUErrorCtrl;

// ʹ�ú�����ӳ����Զ�ӳ��
#define ENUM_VALUE(name) {eGPUErrorType::name, #name},
GENERATE_ENUM_MAP(ENUM_GPU_ERROR_TYPES)
#undef ENUM_VALUE